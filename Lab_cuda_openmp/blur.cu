#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <time.h>

#define BLUR_SIZE 3

using namespace cv;
using namespace std;

__global__
void blurKernel(unsigned char * in, unsigned char * out, int w, int h)
{
  int Col = blockIdx.x * blockDim.x + threadIdx.x;
  int Row = blockIdx.y * blockDim.y + threadIdx.y;
  if (Col < w && Row < h)
  {
    int pixVal = 0;
    int pixels = 0;
    // Get the average of the surrounding 2xBLUR_SIZE x 2xBLUR_SIZE box
    for(int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE+1; ++blurRow)
    {
      for(int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE+1; ++blurCol)
      {
        int curRow = Row + blurRow;
        int curCol = Col + blurCol;
        // Verify we have a valid image pixel
        if(curRow > -1 && curRow < h && curCol > -1 && curCol < w)
        {
          pixVal += in[curRow * w + curCol];
          pixels++; // Keep track of number of pixels in the accumulated total
        }
      }
    }
    // Write our new pixel value out
    out[Row * w + Col] = (unsigned char)(pixVal / pixels);
  }
}

int main(int argc, char** argv) {
  
  clock_t startGPU, endGPU;
  double gpu_time_used;
  if (argc < 3){
    cout << "USO : " << argv[0] << " <entrada.png>" << " <salida.png>" << endl;
    exit(1);
  }

  Mat image;
  // Cargamos la imagen pasada por primer parametro
  image = imread(argv[1], CV_LOAD_IMAGE_COLOR);

  unsigned char *dataRawImage;
  unsigned char *d_dataRawImage, *d_imageOutput;
  unsigned char *h_imageOutput;
  
  Size s = image.size();
  int width = s.width;
  int height = s.height;
  
  int size = sizeof(unsigned char) * width * height * image.channels();

  // Reservar memoria para los objetos en CPU
  dataRawImage = (unsigned char*)malloc(size);
  h_imageOutput = (unsigned char*)malloc(size);

  // Reservar memoria para d_dataRawImage
  hipMalloc((void**)&d_dataRawImage, size);
  // Reservar memoria para la salida de la imegn
  hipMalloc((void**)&d_imageOutput, size);

  // Obtenemos la data
  dataRawImage = image.data;
  startGPU = clock();
  
  // Copiar de dataRawImage a d_dataRawImage
  hipMemcpy(d_dataRawImage, dataRawImage, size, hipMemcpyHostToDevice);
   
  int blockSize = 32;
  dim3 dimBlock(blockSize, blockSize, 1);
  dim3 dimGrid(ceil(width / float(blockSize)), ceil(height / float(blockSize)), 1);
  
  blurKernel<<< dimGrid, dimBlock >>>(d_dataRawImage, d_imageOutput, width, height);
  cout << "Copiando ..." << endl;
  hipMemcpy(h_imageOutput, d_imageOutput, size, hipMemcpyDeviceToHost);
  
  endGPU = clock();
  
  Mat blurImg;
  blurImg.create(height, width, CV_8UC3);
  blurImg.data = h_imageOutput;

  // Guardar la imagen con el segundo parametro de llamada
  imwrite(argv[2], blurImg);

  gpu_time_used = ((double)(endGPU - startGPU)) / CLOCKS_PER_SEC;
  cout << "Tiempo Algoritmo en GPU: " << gpu_time_used << endl;

  hipFree(d_dataRawImage);
  hipFree(d_imageOutput);

  // free(blurImgCPU);
  free(h_imageOutput);

  return 0;
}
