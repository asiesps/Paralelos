
#include <hip/hip_runtime.h>
#include <stdio.h>     

__global__
void matrixVectorKernel(float* A, float* B, float* C, int n)
{
	int i = threadIdx.x + (blockDim.x * blockIdx.x);

	if(i<n){
	C[i] = 0;
	for(int j=0;j<n;j++)
		 C[i] += A[i*n+j] * B[j];
	}
	
}

void matrixVector(float* A, float* B, float* C, int n)
{
	int sizeA = (n*n) * sizeof(float);
	int size =  n * sizeof(float);
	float *d_A,*d_B,*d_C;

	hipMalloc((void**)&d_A,sizeA);
	hipMalloc((void**)&d_B,size);
	hipMalloc((void**)&d_C,size);

	hipMemcpy(d_A,A,sizeA,hipMemcpyHostToDevice);
	hipMemcpy(d_B,B,size,hipMemcpyHostToDevice);

	matrixVectorKernel<<<ceil(n/256.0),256>>>(d_A,d_B,d_C,n);
	hipMemcpy(C,d_C,size,hipMemcpyDeviceToHost);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}

int main(int argc, char* argv[])
{
	int n ;
	n = strtol(argv[1], NULL, 10); 
	
	float *h_a,*h_b,*h_c;
	int size = n*n*sizeof(float);
	int size_v = n*sizeof(float);
	h_a = (float*)malloc(size);
	h_b = (float*)malloc(size_v);
	h_c = (float*)malloc(size_v);

	printf ("Matriz A \n");
	for(int i = 0 ; i < n ; i++){
		for(int j = 0 ; j < n ; j++){
			h_a[i*n +j] = rand() % n + 1;
			printf ("%4.2f \t", h_a[i*n +j]);
		}
		printf ("\n");
   	}

   	printf ("Vector B \n");
	for(int i = 0 ; i < n ; i++){
		h_b[i] = rand() % n + 1;
		printf ("%4.2f \t", h_b[i]);
	}

    matrixVector(h_a, h_b, h_c, n);
    
    printf("Vector Resultado\n");
    for(int i = 0; i < n; i++){
    	printf(" %f \n", h_c[i]);
  	}
  	printf("\n");

   	return 0;
}
