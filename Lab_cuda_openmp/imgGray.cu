#include "hip/hip_runtime.h"
#include <iostream>
#include </usr/include/opencv2/core/core.hpp>
#include </usr/include/opencv2/highgui/highgui.hpp>
#include <time.h>

#define CHANNELS 3

using namespace cv;
using namespace std;

__global__
void imgGrayGPU(unsigned char *imageInput, unsigned char *imageOutput, int width, int height) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  if ((col < width) and (row < height)) {
    int grayOffset = row * width + col;
    int rgbOffset = grayOffset * CHANNELS;

    unsigned char b = imageInput[rgbOffset + 0];
    unsigned char g = imageInput[rgbOffset + 1];
    unsigned char r = imageInput[rgbOffset + 2];

    imageOutput[grayOffset] = 0.21f * r + 0.71f * g + 0.07f * b;
  }
}

int main(int argc, char** argv) {
  
  clock_t startGPU, endGPU;
  double gpu_time_used;
  if (argc < 3){
    cout << "USO : " << argv[0] << " <entrada.png>" << " <salida.png>" << endl;
    exit(1);
  }

  Mat image;
  // Cargamos la imagen pasada por primer parametro
  image = imread(argv[1], CV_LOAD_IMAGE_COLOR);

  unsigned char *dataImage;
  unsigned char *d_dataImage, *d_imageOutput;
  unsigned char *h_imageOutput;
  Size s = image.size();

  int width = s.width;
  int height = s.height;

  int size = sizeof(unsigned char) * width * height * image.channels();
  int sizeGray = sizeof(unsigned char) * width * height;

  dataImage = (unsigned char*)malloc(size);
  h_imageOutput = (unsigned char*)malloc(sizeGray);

  // Reservar memoria para d_dataImage
  hipMalloc((void**)&d_dataImage, size);
// Reservar memoria para la salida de la imegn
  hipMalloc((void**)&d_imageOutput, sizeGray);

  dataImage = image.data;
  startGPU = clock();
  // Copiar de dataImage a d_dataImage
  hipMemcpy(d_dataImage, dataImage, size, hipMemcpyHostToDevice);
    
  int blockSize = 16;
  dim3 dimBlock(blockSize, blockSize, 1);
  dim3 dimGrid(ceil(width / float(blockSize)), ceil(height / float(blockSize)), 1);
  imgGrayGPU<<< dimGrid, dimBlock >>>(d_dataImage, d_imageOutput, width, height);
  hipMemcpy(h_imageOutput, d_imageOutput, sizeGray, hipMemcpyDeviceToHost);
  endGPU = clock();

  Mat grayImg;
  grayImg.create(height, width, CV_8UC1);
  grayImg.data = h_imageOutput;

  // Guardar la imagen con el segundo parametro de llamada
  imwrite(argv[2], grayImg);

  gpu_time_used = ((double)(endGPU - startGPU)) / CLOCKS_PER_SEC;
  cout << "Tiempo Algoritmo en GPU: " << gpu_time_used << endl;

  hipFree(d_dataImage);
  hipFree(d_imageOutput);

  // free(dataImage);
  free(h_imageOutput);

  return 0;
}
