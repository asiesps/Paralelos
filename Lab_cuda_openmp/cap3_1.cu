
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void matrix_addkernel(float* A , float* B , float* C, int n)
{
   int i = threadIdx.x + blockDim.x * blockIdx.x;
   if(i < (n*n))
      C[i] = A[i] + B[i];
}

__global__
void matrix_addkernel_row(float * A , float * B , float * C, int n){
   int i = threadIdx.x + blockDim.x * blockIdx.x;
   if(i < n)
      for(int j =0 ; j < n ; ++j)
         C[i*n+j] = A[i*n+j] + B[i*n+j];

}

__global__
void matrix_addkernel_col(float * A , float * B , float * C, int n){
   int i = threadIdx.x + blockDim.x * blockIdx.x;
   if(i < n)
      for (int j = 0; j < n; ++j)
         C[j*n+i] = A[j*n+i] + B[j*n+i];
}

void sum_matrix(float* A, float* B, float* C, int n)
{
   int size = n*n*sizeof(float);
  
   float *dA, * dB , *dC;
   
   hipMalloc((void**) &dA, size);
   hipMalloc((void**) &dB, size);
   hipMalloc((void**) &dC, size);

   hipMemcpy(dA,A,size,hipMemcpyHostToDevice);
   hipMemcpy(dB,B,size,hipMemcpyHostToDevice);

   //matrix_addkernel<<< ceil((float)n/256.0), 256>>>(dA, dB, dC,n);
   //matrix_addkernel_row<<< ceil((float)n/256.0), 256>>>(dA, dB, dC,n);
   matrix_addkernel_col<<< ceil((float)n/256.0), 256>>>(dA, dB, dC,n);
   
   hipMemcpy(C,dC,size,hipMemcpyDeviceToHost);

   hipFree(dA);
   hipFree(dB);
   hipFree(dC);
   
}

int main(int argc, char* argv[])
{
   int n ;
   n = strtol(argv[1], NULL, 10); 
   
   float * h_a, * h_b, * h_c;
   int size = n*n*sizeof(float);
   h_a = (float*)malloc(size);
   h_b = (float*)malloc(size);
   h_c = (float*)malloc(size);

   printf ("Matriz A \n");
   for(int i = 0 ; i < n ; i++){
      for(int j = 0 ; j < n ; j++){
         h_a[i*n +j] = rand() % 100;
         printf ("%4.2f \t", h_a[i*n +j]);
      }
      printf ("\n");
   }
   printf ("Matriz B \n");
   for(int i = 0 ; i < n ; ++i){
      for(int j = 0 ; j < n ; ++j){
         h_b[i*n +j] = rand() % 100;
         printf ("%4.2f \t", h_b[i*n +j]);
      }
      printf ("\n");
   }
   
   sum_matrix(h_a, h_b, h_c, n);

   printf("RESULTADOS :\n");   
   for(int i = 0 ; i < n ; ++i){
      for(int j = 0 ; j < n ; ++j){
         printf ("%4.2f \t", h_c[j*n +i]);
      }
      printf ("\n");
   }
   
}
