#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

// #define Height 32768
// #define Width 32768
int Width = 10240  ;
int Height = 10240 ;
using namespace std;
float aleatorio(){
  return random()/((double) RAND_MAX);
}

void llenar_matriz(float* M) {
  for(int i = 0; i < Height; i++) {
    for(int j = 0; j < Width; j++) {
      M[i*Width + j] = 1;
    }
  }
}

void Matrix_mult(float* A, float* B,float* C) {
  float tmp = 0;
  for(int i = 0; i < Height; i++) {
    for(int j = 0; j < Width; j++) {
      tmp = 0;
      for(int k=0; k < Width; k++)
        tmp += A[i * Width + k] * B[k * Width + j];
     C[i * Width + j] = tmp;
    }
  }

}

void print_matriz(float* v) {
  for(int i = 0; i < Height; i++){
    for(int j = 0; j < Width; j++){
      cout << v[i * Width + j] << " ";
    }
    cout << endl;
  }
}

__global__ 
void MatrixMulKernel(float* M, float* N, float* P,int Width) {
  // Calculate the row index of the P element and M
  int Row = blockIdx.y*blockDim.y+threadIdx.y;
  // Calculate the column index of P and N
  int Col = blockIdx.x*blockDim.x+threadIdx.x;

  if ((Row < Width) && (Col < Width)) {
    float Pvalue = 0;
    // each thread computes one element of the block sub-matrix
    for (int k = 0; k < Width; ++k) {
      Pvalue += M[Row*Width+k]*N[k*Width+Col];
    }
    P[Row*Width+Col] = Pvalue;
  }
}

int main() {
  clock_t inicio,fin;
  double tiempo_cpu;

  // Separo espacio de memoria para las variables en host
  float* A = new float[Height*Width];
  float* B = new float[Height*Width];
  float* C = new float[Height*Width];
  float* CC = new float[Height*Width];

  llenar_matriz(A);
  llenar_matriz(B);
// ========================================================
  // inicio = clock();
  // // print_matriz(A);
  // // print_matriz(B);
  // Matrix_mult(A, B, C);
  // // print_matriz(C);

  // fin = clock();
  // tiempo_cpu = ((double)(fin - inicio)) / CLOCKS_PER_SEC;
  // cout << "Tiempo en CPU : " << tiempo_cpu << endl;

// ========================================================
  float *d_A, *d_B, *d_C;
  float size_block = 64;

  dim3 dimBlock(size_block, size_block);
  dim3 dimGrid(ceil(Width / float(size_block)), ceil(Height / float(size_block)), 1);
  
  int size = sizeof(float)*Height*Width;

  hipMalloc((void**)&d_A, size);
  hipMalloc((void**)&d_B, size);
  hipMalloc((void**)&d_C, size);

  inicio = clock();
  hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

  MatrixMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, Width);
  hipMemcpy(CC, d_C, size, hipMemcpyDeviceToHost);
  // print_matriz(CC);
  fin = clock();
  tiempo_cpu = ((double)(fin - inicio))/CLOCKS_PER_SEC;
  cout << "Tiempo en GPU : " << tiempo_cpu << endl;

  delete A; delete B;
  delete C; delete CC;

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}
