#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;
#define TILE_WIDTH 32
// #define Height 1024
// #define Width 1024
int Width = 1024 ;
int Height = 1024 ;
using namespace std;
float aleatorio(){
  return random()/((double) RAND_MAX);
}

void llenar_matriz(float* M) {
  for(int i = 0; i < Height; i++) {
    for(int j = 0; j < Width; j++) {
      M[i * Width + j] = 1;
    }
  }
}

void Matrix_mult(float* A, float* B,float* C) {
  float tmp = 0;
  for(int i = 0; i < Height; i++) {
    for(int j = 0; j < Width; j++) {
      tmp = 0;
      for(int k=0; k < Width; k++)
        tmp += A[i * Width + k] * B[k * Width + j];
     C[i * Width + j] = tmp;
    }
  }

}

void print_matriz(float* v) {
  for(int i = 0; i < Height; i++){
    for(int j = 0; j < Width; j++){
      cout << v[i * Width + j] << " ";
    }
    cout << endl;
  }
}

__global__
void MatrixMulKernel_Gran(float* M, float* N, float* P,int Width)
{
  __shared__ float ds_M[TILE_WIDTH][TILE_WIDTH];
  __shared__ float ds_N[TILE_WIDTH][TILE_WIDTH];
  __shared__ float ds_N_N[TILE_WIDTH][TILE_WIDTH];
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int Row = by * TILE_WIDTH + ty;
  int Col = bx*2* TILE_WIDTH + tx;

  float Pvalue = 0;
  float val_2 = 0;
  // Loop over the M and N tiles required to compute the P element
  if((Row < Width) && (Col < Width)){
    for (int ph = 0; ph < Width/TILE_WIDTH; ++ph){

      // Collaborative loading of M and N tiles into shared memory
      ds_M[ty][tx] = M[Row*Width + ph*TILE_WIDTH+tx];    //ph=indice del azulejo
      ds_N[ty][tx] = N[(ph*TILE_WIDTH+ty)*Width + Col];
      ds_N_N[ty][tx] = N[(ph*TILE_WIDTH+ty)*Width + (Col+TILE_WIDTH)];
      __syncthreads();

      for (int i = 0; i < TILE_WIDTH; ++i)
      {
          Pvalue += ds_M[ty][i] * ds_N[i][tx];
          val_2 += ds_M[ty][i] * ds_N_N[i][tx];
      }

      __syncthreads();
    }
    P[Row*Width+Col] = Pvalue;
    P[(Row*Width+Col)+TILE_WIDTH] = val_2;
  }
}

int main() {
  clock_t inicio,fin;
  double tiempo_cpu;

  // Separo espacio de memoria para las variables en host
  float* A = new float[Height*Width];
  float* B = new float[Height*Width];
  float* C = new float[Height*Width];
  float* CC = new float[Height*Width];

  llenar_matriz(A);
  llenar_matriz(B);
// ========================================================
  // inicio = clock();
  // print_matriz(A);
  // print_matriz(B);
  cout << endl;
  // Matrix_mult(A, B, C);
  // // print_matriz(C);

  // fin = clock();
  // tiempo_cpu = ((double)(fin - inicio)) / CLOCKS_PER_SEC;
  // cout << "Tiempo en CPU : " << tiempo_cpu << endl;

// ========================================================

  float *d_A, *d_B, *d_C;
  float blockSize = TILE_WIDTH;

  dim3 dimBlock(blockSize, blockSize);
  dim3 dimGrid(ceil(Width/float(blockSize)), ceil(Height/float(blockSize)), 1);
  
  int size = sizeof(float)*Height*Width;

  hipMalloc((void**)&d_A, size);
  hipMalloc((void**)&d_B, size);
  hipMalloc((void**)&d_C, size);

  inicio = clock();
  hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

  MatrixMulKernel_Gran<<<dimGrid, dimBlock>>>(d_A, d_B, d_C,Width);
  hipMemcpy(CC, d_C, size, hipMemcpyDeviceToHost);

  fin = clock();
  tiempo_cpu = ((double)(fin - inicio))/CLOCKS_PER_SEC;
  cout << "Tiempo en GPU : " << tiempo_cpu << endl;
  // print_matriz(CC);
  delete A; delete B;
  delete C; delete CC;

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}
