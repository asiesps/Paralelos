#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;
#define TILE_WIDTH 64
// #define Height 1024
// #define Width 1024
int Width = 1024 ;
int Height = 1024 ;
using namespace std;
float aleatorio(){
  return random()/((double) RAND_MAX);
}

void llenar_matriz(float* M) {
  for(int i = 0; i < Height; i++) {
    for(int j = 0; j < Width; j++) {
      M[i * Width + j] = 1;
    }
  }
}

void Matrix_mult(float* A, float* B,float* C) {
  float tmp = 0;
  for(int i = 0; i < Height; i++) {
    for(int j = 0; j < Width; j++) {
      tmp = 0;
      for(int k=0; k < Width; k++)
        tmp += A[i * Width + k] * B[k * Width + j];
     C[i * Width + j] = tmp;
    }
  }

}

void print_matriz(float* v) {
  for(int i = 0; i < Height; i++){
    for(int j = 0; j < Width; j++){
      cout << v[i * Width + j] << " ";
    }
    cout << endl;
  }
}

__global__
void MatrixMulKernel_Tailed(float* M, float* N, float* P,int Width){
  __shared__ float ds_M[TILE_WIDTH][TILE_WIDTH];
  __shared__ float ds_N[TILE_WIDTH][TILE_WIDTH];
  int bx = blockIdx.x; 
  int by = blockIdx.y;
  int tx = threadIdx.x; 
  int ty = threadIdx.y;

  int Row = by * TILE_WIDTH + ty;
  int Col = bx * TILE_WIDTH + tx;

  float Pvalue = 0;
  // Loop over the M and N tiles required to compute the P element
  for (int ph = 0; ph < Width/TILE_WIDTH; ++ph) {    

    // ph representa el índice de mosaico que se encuentra, 
    // por lo que ph * T_Wle dará el índice x-global del primer thread en un bloque. 
    // Dentro de cada bloque, los hilos tienen sus propios índices y por lo tanto
    // es necesario añadir el x-índice del thread dentro de ese bloque. 
    // Por lo que tiene p * T_W + tx.
    // Collaborative loading of M and N tiles into shared memory
    ds_M[ty][tx] = M[Row*Width + ph*TILE_WIDTH+tx];    //ph=indice del azulejo
    ds_N[ty][tx] = N[(ph*TILE_WIDTH+ty)*Width + Col];  

    __syncthreads();

    for (int i = 0; i < TILE_WIDTH; ++i)
      Pvalue += ds_M[ty][i] * ds_N[i][tx];

    __syncthreads();
  }
  P[Row*Width+Col] = Pvalue;
}

int main() {
  clock_t inicio,fin;
  double tiempo_cpu;

  // Separo espacio de memoria para las variables en host
  float* A = new float[Height*Width];
  float* B = new float[Height*Width];
  float* C = new float[Height*Width];
  float* CC = new float[Height*Width];

  llenar_matriz(A);
  llenar_matriz(B);
// ========================================================
  // inicio = clock();
  // print_matriz(A);
  // print_matriz(B);
  cout << endl;
  // Matrix_mult(A, B, C);
  // // print_matriz(C);

  // fin = clock();
  // tiempo_cpu = ((double)(fin - inicio)) / CLOCKS_PER_SEC;
  // cout << "Tiempo en CPU : " << tiempo_cpu << endl;

// ========================================================

  float *d_A, *d_B, *d_C;
  float blockSize = TILE_WIDTH;

  dim3 dimBlock(blockSize, blockSize);
  dim3 dimGrid(ceil(Width/float(blockSize)), ceil(Height/float(blockSize)), 1);
  
  int size = sizeof(float)*Height*Width;

  hipMalloc((void**)&d_A, size);
  hipMalloc((void**)&d_B, size);
  hipMalloc((void**)&d_C, size);

  inicio = clock();
  hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

  MatrixMulKernel_Tailed<<<dimGrid, dimBlock>>>(d_A, d_B, d_C,Width);
  hipMemcpy(CC, d_C, size, hipMemcpyDeviceToHost);

  fin = clock();
  tiempo_cpu = ((double)(fin - inicio))/CLOCKS_PER_SEC;
  cout << "Tiempo en GPU : " << tiempo_cpu << endl;
  print_matriz(CC);
  delete A; delete B;
  delete C; delete CC;

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}
